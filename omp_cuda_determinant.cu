#include <stdio.h>
#include <stdlib.h>
#include <dirent.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <omp.h>


#define LINE_BUFFER_SIZE 200000


// prints the given square matrix
void print_matrix(double *matrix, int n) {
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			printf("%lf ", matrix[i * n + j]);
		}
		printf("\n");
	}
}

int DecomposeLU(double *matrix, int n) {

	double epsilon = 0.000000001;
	int swap = 1;

	hipblasStatus_t hipblasStatus_t;

	// creating cuBLAS handle
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// creating CUDA stream
	hipStream_t stream;
	hipStreamCreate(&stream);

	// setting the stream
	hipblasSetStream(handle, stream);


	for (int k = 0; k < n - 1; k++) {

		// finding the pivot row
		int pivotRow;
		hipblasStatus_t = hipblasIdamax(handle, n - k, matrix + k + k * n, 1, &pivotRow);
		if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
			printf ("hipblasIdamax failed!");
			exit(-1);
		}

		pivotRow += k - 1;
		int kp1 = k + 1;

		// getting the pivot row to the top
		if (pivotRow != k) {
			swap *= -1;
			hipblasStatus_t = hipblasDswap(handle, n, matrix + pivotRow, n, matrix + k, n);
			if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
				printf ("hipblasDswap failed!");
				exit(-1);
			}
		}

		// checking if we got 0 on the diagonal entry
		double valcheck;
		hipblasStatus_t = hipblasGetVector(1, sizeof(double), matrix + k + k * n, 1, &valcheck, 1);
		if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
			printf ("hipblasGetVector failed!");
			exit(-1);
		}


		if (fabs(valcheck) < epsilon)
		   return swap;

		// finding partial L and U
		if (kp1 < n) {
			const double alpha = 1.0f / valcheck;
			hipblasStatus_t = hipblasDscal(handle, n - kp1, &alpha , matrix + kp1 + k * n, 1);
			if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
				printf ("hipblasDscal failed!");
				exit(-1);
			}
		}

		if (kp1 < n) {
			const double alpha = -1.0f;
			hipblasStatus_t = hipblasDger(handle, n - kp1, n - kp1, &alpha, matrix + kp1 + k * n, 1, matrix + k + kp1 * n, n, matrix + kp1 * n + kp1, n);
			if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
				printf ("hipblasDger failed!");
				exit(-1);
			}
		}

	}

	// destroying the stream
	hipStreamDestroy(stream);

	return swap;

}

// calculates the determinant of input matrix, using Row-Reduction algorithm
double determinant(double *matrix, int n) {

	// allocating memory for device matrix
	double *d_matrix;

	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void**) &d_matrix, n * n * sizeof(double));
	if (cudaStatus != hipSuccess) {
		printf("allocating device memory for the matrix failed!");
		exit(-1);
	}

	// copying the h_matrix into d_matrix
	cudaStatus = hipMemcpy(d_matrix, matrix, n * n * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("copying matrix into d_matrix failed!");
		exit(-1);
	}

	// LU decomposition
	int swap = DecomposeLU(d_matrix, n);

	cudaStatus = hipMemcpy(matrix, d_matrix, n * n * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("getting matrix from device failed!");
		exit(-1);
	}

	double det = 1.0;
	for (int i = 0; i < n; i++) {
		det *= matrix[i * n + i];
	}

	// taking the number of swaps into consideration
	det *= swap;

	return det;
}


void processFile(char *fileName) {

	// making input file path
	char inputFilePath[40] = "data_in/";
	strcat(inputFilePath, fileName);

	// opening the input file
	FILE *ifp;
	ifp = fopen(inputFilePath, "r");

	// making input file path
	char outputFilePath[40] = "data_out/";
	strcat(outputFilePath, fileName);

	// creating the empty output file
	FILE *ofp;
	ofp = fopen(outputFilePath, "w");

	// line buffer
	char line[LINE_BUFFER_SIZE];

	// number of matrices read so far
	int m = 0;

	while (1) {

		// reading '\n' between the matrices
		if (m > 0) {
			char *ret = fgets(line, LINE_BUFFER_SIZE, ifp);
			// checking if we've reached the end of the file
			if (ret == NULL)
				break;
		}

		// reading the first line of the current matrix
		char *ret = fgets(line, LINE_BUFFER_SIZE, ifp);

		// checking if we've reached the end of the file
		if (ret == NULL)
			break;

		// finding the size of the current matrix by counting the ' ' characters
		int n = 0;
		for (int i = 0; i < LINE_BUFFER_SIZE; i++) {
			if (line[i] == '\n')
				break;
			if (line[i] == ' ')
				n++;
		}

		// allocating memory for the current matrix
		double *matrix = (double*) malloc(n * n * sizeof(double));

		// storing the first line elements into the matrix
		char *savePtr;
		char *token = strtok_r(line, " ", &savePtr);
		for (int i = 0; i < n; i++) {
			double d;
			sscanf(token, "%lf", &d);
			matrix[i] = d;
			token = strtok_r(NULL, " ", &savePtr);
		}

		// reading the rest of the file
		for (int i = 1; i < n; i++) {
			fgets(line, LINE_BUFFER_SIZE, ifp);

			char * token = strtok_r(line, " ", &savePtr);
			for (int j = 0; j < n; j++) {
				double d = 1.0;
				sscanf(token, "%lf", &d);
				matrix[i * n + j] = d;
				token = strtok_r(NULL, " ", &savePtr);
			}
		}

		// calculating the determinant
		double det = determinant(matrix, n);

		// writing the result in output file
		fprintf(ofp, "%lf\n", det);

		free(matrix);
		m++;

	}

	// closing the files
	fclose(ifp);
	fclose(ofp);

}

int main() {

	double startTime = omp_get_wtime();

	#pragma omp parallel
	{
		#pragma omp single nowait
		{
			// listing all the input files in 'data_in' folder
			DIR *d;
			struct dirent *dir;
			d = opendir("data_in");

			if (d) {

				// for every file in 'data_in'
				while ((dir = readdir(d)) != NULL) {
					if (strcmp(dir->d_name, ".") != 0 && strcmp(dir->d_name, "..") != 0) {

						char fileName[30] = "";
						strcat(fileName, dir->d_name);

						// create a task for the file to be done by a thread
						#pragma omp task firstprivate(fileName)
						processFile(fileName);
					}
				}
				closedir(d);
			}
		}
	}

	printf("%fs\n", omp_get_wtime() - startTime);
}
